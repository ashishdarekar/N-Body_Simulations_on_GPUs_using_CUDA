#include "hip/hip_runtime.h"
// implementation of CUDA code for N Body Simulations written by Ashish and Shubham 
#include <math.h>
#include <ostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <random>
#include <fenv.h>
#include <iostream>
#include <fstream>
#include <omp.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include "constant.h"


// cuda error checking
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line); 
        if (abort) exit(code);
    }
}

// Function to calculate the force and hence acceleration [24 FLOPS]
__device__ float3 acceleration(float4 posi, float4 posj, float3 F){
            
            // r_ij  [6 FLOPS]
            float dx = (posj.x - posi.x)*TO_METERS;
            float dy = (posj.y - posi.y)*TO_METERS;
            float dz = (posj.z - posi.z)*TO_METERS;
            
            // [6 Flops]
            float distSqr = dx*dx + dy*dy + dz*dz + (SOFTENINGSQ); //Numerator of the equation
            
            // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
            float invDist = 1.0f / sqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist; 

            //Sumation of the forces on i particle by j particle (check addition of masses is fine)
            //[ 2 + 6 = 8FLOPS]
            float gdist = G * (posj.w) * invDist3 ;

            F.x += gdist * dx ;
            F.y += gdist * dy ;
            F.z += gdist * dz ;
            return F;
}

//Kernel for Force calculation on each body. 
//FOR DISC in AU
//Tile computation 
__global__ void discbodyinteraction(float4 *pos, float4 *vel, float dt, int num_body, int cuda_Blocksize)
{
  //indexing in CUDA to capture correct body
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  extern __shared__ float4 shposition[]; //making shared memory of that threadblock
  if(i < num_body)
  {
    
    for (int tile = 0; tile < gridDim.x; tile++) //calculations of tiles (in a thradbloack) in a sequestial way
    { 
        //Tile preparation and copy of a data from global memory to share memory of threadblock
        float3 F;
        F.x = 0.0f;
        F.y = 0.0f;
        F.z = 0.0f;
        // float Fx = 0.0f; 
        // float Fy = 0.0f; 
        // float Fz = 0.0f;
        shposition[threadIdx.x] = pos[tile * blockDim.x + threadIdx.x];
        __syncthreads(); //To make sure all threads in a tile are copied data propely from Global memory before starting calculation
          
        //Same force calculation as that in serial
        for (int j = 0; j < cuda_Blocksize; ) 
        {
          F = acceleration(pos[i], shposition[j], F);
          j += 1;

          #if LOOP_UNROLL > 1
          F = acceleration(pos[i], shposition[j], F);
          j += 1;
          // printf("\nLoop unroll for 1!\n");
          #endif

          #if LOOP_UNROLL > 2
          F = acceleration(pos[i], shposition[j], F);
          j += 1;
          F = acceleration(pos[i], shposition[j], F);
          j += 1;
          // printf("\nLoop unroll for 2!\n");
          #endif

          #if LOOP_UNROLL > 4
          F = acceleration(pos[i], shposition[j], F);
          j += 1;
          F = acceleration(pos[i], shposition[j], F);
          j += 1;
          F = acceleration(pos[i], shposition[j], F);
          j += 1;
          F = acceleration(pos[i], shposition[j], F);
          j += 1;
          // printf("\nLoop unroll for 4!\n");
          #endif
        }
        __syncthreads(); //To make sure all threads in a tile are done with the calculation before proceed with the velocity updation

        //Acceleration and new velocitites of particle i [6 FLOPS]
        vel[i].x += ( (dt*F.x) ); 
        vel[i].y += ( (dt*F.y) ); 
        vel[i].z += ( (dt*F.z));
    }
    //Position update [1+6 = 7 FLOPS]
      pos[i].x += ( (vel[i].x)*dt / TO_METERS );
      pos[i].y += ( (vel[i].y)*dt / TO_METERS );
      pos[i].z += ( (vel[i].z)*dt / TO_METERS );
  }
}

__global__ void initializer(float4 *pos, float4 *vel, const int num_body){
  hiprandState state;
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  #if TWO_STAR
  if (i==0){
    pos[i].x = TWO_STAR_ORBIT;
    pos[i].y = 0.0;
    pos[i].z = 0.0;
    pos[i].w = 100*SOLAR_MASS;
    vel[i].x = 0.0;
    vel[i].y = -pow(((G*(SOLAR_MASS + EXTRA_MASS*SOLAR_MASS))/ (TWO_STAR_ORBIT*TO_METERS)), 0.5);
    vel[i].z = 0.0;
  }
  else if (i==1){
    pos[i].x = -TWO_STAR_ORBIT;
    pos[i].y = 0.0;
    pos[i].z = 0.0;
    pos[i].w = 100*SOLAR_MASS;
    vel[i].x = 0.0;
    vel[i].y = pow(((G*(SOLAR_MASS + EXTRA_MASS*SOLAR_MASS))/ (TWO_STAR_ORBIT*TO_METERS)), 0.5);
    vel[i].z = 0.0;
  }
  #else
  if (i==0){
    pos[i].x = 0.0;
    pos[i].y = 0.0;
    pos[i].z = 0.0;
    pos[i].w = SOLAR_MASS;
    vel[i].x = 0.0;
    vel[i].y = 0.0;
    vel[i].z = 0.0;
  }
  #endif
  else{
    float factor;
    
    #if TWO_STAR
    factor = 1.0;
    #else
    factor = 1.0;
    #endif
    
    float angle;
    float radius, randRadius;
    float velocity, randHeight;
    velocity = 0.67*sqrt((G*SOLAR_MASS)/(4*BINARY_SEPARATION * TO_METERS));
    // reference article https://stackoverflow.com/questions/18501081/generating-random-number-within-cuda-kernel-in-a-varying-range
    hiprand_init(clock64(), i, 0, &state);
    angle = hiprand_uniform(&state)*(2.0*PI - 0.0);
    hiprand_init(clock64(), i, 0, &state);
    randRadius = hiprand_uniform(&state)*(SYSTEM_SIZE - INNER_BOUND) + INNER_BOUND;
    hiprand_init(clock64(), i, 0, &state);
    randHeight = hiprand_uniform(&state)*(SYSTEM_THICKNESS);

    radius = sqrtf(SYSTEM_SIZE)*sqrtf(randRadius);
		velocity = pow((G*((200*SOLAR_MASS)+1*(EXTRA_MASS*SOLAR_MASS))/ (radius*TO_METERS)), 0.5);

    pos[i].x = radius*cos(angle);
    pos[i].y = radius*sin(angle);
    pos[i].z = randHeight-SYSTEM_THICKNESS/2.0;
    pos[i].w = (EXTRA_MASS*SOLAR_MASS)/num_body;
    vel[i].x = velocity*sin(angle);
    vel[i].y = -velocity*cos(angle);
    vel[i].z = 0.0;
  }
}

//Visualisation using VTk
void write_vtkFile(const char *szProblem, int timeStepNumber, int num_body, cudaBody *p ) 
{
  char szFileName[80];
  FILE *fp=NULL;
  sprintf( szFileName, "%s.%i.vtk", szProblem, timeStepNumber );
  fp = fopen( szFileName, "w");
  if( fp == NULL )		       
  {
    char szBuff[80];
    sprintf( szBuff, "Failed to open %s", szFileName );
    //ERROR( szBuff );
    return;
  }

  // Write VTK Header
  fprintf(fp,"# vtk DataFile Version 2.0\n");
  fprintf(fp,"generated for CUDA Seminar output (written by Ashish and Shubham) \n");
  fprintf(fp,"ASCII\n");
  fprintf(fp,"DATASET POLYDATA\n");
  fprintf(fp,"POINTS %i float\n", num_body);

  // Write VTK point Coordinator
  for( int i = 0; i < num_body; i++ ) 
  {
    fprintf(fp, "%f %f %f\n",(p->position[i].x), (p->position[i].y), (p->position[i].z));
  }
	
  if( fclose(fp) )
  {
    char szBuff[80];
    sprintf( szBuff, "Failed to close %s", szFileName );
    //ERROR( szBuff );
  }
}

//Function to calculate the Performance in GFLOPS
double computePerf(int cuda_numblocks, float timeseconds, const int iterations, int num_body)
{
    int flopsPerInteraction=0;
    flopsPerInteraction = ( num_body * num_body * 24 ) + ( num_body *  (num_body/cuda_numblocks) * 6 ) + ( num_body * 7);
    
    double total_operations = iterations * (double)flopsPerInteraction ; 
    double gflops = 1e-9 * ((double)total_operations) / timeseconds  ;
    
    return gflops;
}


int main(int argc, char* argv[])
{
  int numBodies;
  int cuda_blocksize;

  if (argc>=2)
  {
    if (std::string(argv[1]) == "-s") 
    {      
      numBodies = std::stoi(argv[2]);  
    } 
    else
    {
      fprintf(stderr, "Wrong argument - %s\n", argv[1]);
      exit(EXIT_FAILURE);
    }
    if (std::string(argv[3]) == "-b"){
      cuda_blocksize = std::stoi(argv[4]);
    }
    else
    {
      fprintf(stderr, "Wrong argument - %s\n", argv[4]);
      exit(EXIT_FAILURE);
    }
  }
  else
  {
    fprintf(stderr, "Usage:./self_cuda_tile -s <numBodies> -b <blocksize>\n");
    exit(EXIT_FAILURE);
  }
  //To save all the visualisation files in the folder
  char sol_directory[100];
  char sol_folder[100];
  struct stat st;

#if VTKWRITE

  sprintf(sol_folder,"cuda_solution_%s","cuda_seminar");
  if(stat(sol_folder,&st)==-1)
    mkdir(sol_folder,0700);

  sprintf(sol_directory,"cuda_solution_%s/sol","cuda_seminar");

#endif

  //Memory allocation on the HOST
  int size_mem = numBodies * 2 * sizeof(float4);
  float4 *h_buf = (float4*) malloc(size_mem);
  cudaBody ph = {h_buf, h_buf + numBodies }; //2 arrays, positions and velocity arrays, so ph pointing to the h_buf

  //Memory allocation on the DEVICE
  float4 *d_buf;
  hipMalloc((void**) &d_buf, size_mem );
  cudaBody pd = { d_buf, d_buf + numBodies }; 

  int cuda_numblocks = (numBodies + cuda_blocksize - 1) / cuda_blocksize;  

  //Time Iterations 
  clock_t startTime = clock();
  dim3 grid = dim3(cuda_numblocks, 1, 1);
  dim3 threads = dim3(cuda_blocksize, 1, 1);

  initializer<<<grid, threads>>>(pd.position, pd.velocity, numBodies);
  
  hipMemcpy(h_buf, d_buf, size_mem , hipMemcpyDeviceToHost);
  write_vtkFile(sol_directory, 0, numBodies, &ph);

  // for timining
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);//Time Begin
  
  for (int iter=1; iter<=numIters; iter++)
  {
    //Copy data from Host to Device, buffer copying 
    discbodyinteraction<<<grid, threads, cuda_blocksize*sizeof(float4)>>>( pd.position, pd.velocity, dt, numBodies, cuda_blocksize);
  } 

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("\nElapsed time for Computation:  %10.5f sec \n", time/1000.0f);
  double gflops=0.0;
  gflops = computePerf(cuda_numblocks, time/1000.0f, numIters, numBodies);
  
  printf("\nCUDA N-Body (%d bodies): %f GFLOP/s\n", numBodies, gflops);

  hipMemcpy(h_buf, d_buf, size_mem , hipMemcpyDeviceToHost);
  write_vtkFile(sol_directory, numIters, numBodies, &ph);

  free(h_buf);
  hipFree(d_buf);
}

